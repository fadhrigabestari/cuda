
#include "hip/hip_runtime.h"

#include <iterator>
#include <iostream>
#include <algorithm>
#include <fstream>
#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <stdint.h>

using namespace std;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

void printToFile(int* arr, int n) {
	ofstream fstream;
	fstream.open("data/output");
	for (int i = 0; i < n; i++) {
		fstream << arr[i] << endl;
	}
}

void printArr(int* arr, int n) {
	for (int i = 0; i < n; i++) {
		cout << arr[i] << endl;
	}
}

void rng(int* arr, int n) {
	int seed = 13516154;
	srand(seed);
	for (long i = 0; i < n; i++) {
		arr[i] = (int)rand();
	}
}

// parallel radix sort
// get specific bit at index = idx
__global__ void generateFlag(int* flag, int* arr, int n, int idx) {

	// parallel
	for (int i = 0; i < n; i++) {
		if ((arr[i] >> idx) & 1 == 1) {
			flag[i] = 1;
		}
		else {
			flag[i] = 0;
		}
	}
}

// create I-down array
int* generateIDown(int* flag, int n) {
	int* iDown = (int*)malloc(n * sizeof(int));
	int val = 0;

	iDown[0] = val;
	for (int i = 1; i < n; i++) {
		if (flag[i - 1] == 0) {
			val++;
		}
		iDown[i] = val;
	}
	return iDown;
}

// create I-up array
int* generateIUp(int* flag, int n) {
	int* iUp = (int*)malloc(n * sizeof(int));
	int val = n - 1;

	iUp[n - 1] = val;
	for (int i = n - 2; i >= 0; i--) {
		if (flag[i + 1] == 1) {
			val--;
		}
		iUp[i] = val;
	}
	return iUp;
}

int* generateShouldIndex(int* flag, int* iDown, int* iUp, int n) {
	int* shouldIndex = (int*)malloc(n * sizeof(int));

	// parallel
	for (int i = 0; i < n; i++) {
		if (flag[i] == 0) {
			shouldIndex[i] = iDown[i];
		}
		else {
			shouldIndex[i] = iUp[i];
		}
	}
	return shouldIndex;
}

void permute(int* arr, int* flag, int* iDown, int* iUp, int n) {
	int* shouldArr = (int*)malloc(n * sizeof(int));

	int* shouldIndex = generateShouldIndex(flag, iDown, iUp, n);

	// parallel
	for (int i = 0; i < n; i++) {
		shouldArr[shouldIndex[i]] = arr[i];
	}

	// parallel
	for (int i = 0; i < n; i++) {
		arr[i] = shouldArr[i];
	}
}

void split(int* arr, int n, int idx) {
	int blockSize = 256;
	int numBlocks = (n + blockSize - 1) / blockSize;

  int* h_flag = (int*)malloc(n * sizeof(int));
  int* d_flag;

  int* d_arr;

  hipMalloc(&d_flag, n * sizeof(int));
  hipMalloc(&d_arr, n * sizeof(int));

  hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);

	generateFlag<<<numBlocks,blockSize>>>(d_flag, d_arr, n, idx);
  hipDeviceSynchronize();
  
  hipMemcpy(h_flag, d_flag, n * sizeof(int), hipMemcpyDeviceToHost);

	int* iDown = generateIDown(h_flag, n);
	int* iUp = generateIUp(h_flag, n);

  permute(arr, h_flag, iDown, iUp, n);
  cout << "progress: ";
  printArr(arr, n);
}


void radixSort(int* arr, int n) {
	int idx = 0;

	for (int i = 0; i < 32; i++) {
		split(arr, n, i);
	}
}

int main(int argc, char** argv)
{
	int n = 2;

	//if (argc != 2) {
		//cout << "Wrong input" << endl;
		//cout << "./radix_sort <N>" << endl;
		//exit(0);
	//}
	//else {
		//n = int(argv[1]);
	//}


	int * arr = (int*)malloc(n * sizeof(int));

	rng(arr, n);
	printArr(arr,n);

	clock_t beginTime = clock();
  radixSort(arr, n);
  clock_t endTime = clock();	
  printArr(arr, n);

	double elapsedTime = (double)endTime - beginTime / CLOCKS_PER_SEC;

	cout << "Parallel Radix Sort Time: " << elapsedTime << endl;
	cout << endl;

    return 0;
}
